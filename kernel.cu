#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fcntl.h>

#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <cstring>  // For memcpy

#include <windows.h>
#define SHM_NAME "/shared_mem"


const int CTX_SIZE_BYTES = 8 * 20; // 160
const int KEY_SIZE_BYTES = 32;
const int HASH_NO_SIG_SIZE_BYTES = 32;
const int TOTAL_BYTES_SEND = CTX_SIZE_BYTES + KEY_SIZE_BYTES + HASH_NO_SIG_SIZE_BYTES;



/*************************** HEADER FILES ***************************/
#include <stdlib.h>
#include <memory.h>

/****************************** MACROS ******************************/
#define SHA256_BLOCK_SIZE 32            // SHA256 outputs a 32 byte digest

/**************************** DATA TYPES ****************************/
typedef struct {
    BYTE data[64];
    WORD datalen;
    unsigned long long bitlen;
    WORD state[8];
} CUDA_SHA256_CTX;


struct SharedData {
    volatile uint64_t nonce;
    volatile uint8_t data[TOTAL_BYTES_SEND];      // Buffer to send data
};


// Define a struct to represent a uint256 (256-bit integer)
struct uint256 {
    uint64_t data[4];  // Array to hold four 64-bit parts
};

int main(int argc, char* argv[]) {

    int gpu_num = 0; // default
    if (argc == 2)
    {
        gpu_num = (int)atoi(argv[1]);
    }


    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    // Initialize the CUDA driver API
    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);

    // Load the PTX module
    hipError_t res = hipModuleLoad(&cuModule, "kernel.ptx");
    if (res != hipSuccess) {
        std::cerr << "Failed to load PTX\n";
        return 1;
    }

    // Get the kernel function
    // Use the mangled name
    // cat kernel.ptx  | grep cuda_miner
    // .globl	_Z10cuda_minerPhS_S_S_S_
    hipModuleGetFunction(&cuFunction, cuModule, "_Z10cuda_minerPhS_S_S_S_");


    const int CTX_SIZE_BYTES = 8 * 20; // 160
    const int KEY_SIZE_BYTES = 32;
    const int HASH_NO_SIG_SIZE_BYTES = 32;
    const int TOTAL_BYTES_SEND = CTX_SIZE_BYTES + KEY_SIZE_BYTES + HASH_NO_SIG_SIZE_BYTES;

    const int NONCE_SIZE_BYTES = 8;


    //uint8_t *d_gpu_num;
    // Allocate pinned host memory  
    void* h_gpu_num;
    hipHostAlloc(&h_gpu_num, 1, hipHostMallocPortable);
    *static_cast<uint8_t*>(h_gpu_num) = static_cast<uint8_t>(gpu_num);

    //////////////////////STAGE2==================

    //uint8_t *d_ctx_data;
    uint8_t* h_ctx_data = new uint8_t[CTX_SIZE_BYTES];

    //uint8_t *d_key_data;
    uint8_t* h_key_data = new uint8_t[KEY_SIZE_BYTES];

    //uint8_t *d_hash_no_sig_data;
    uint8_t* h_hash_no_sig_data = new uint8_t[HASH_NO_SIG_SIZE_BYTES];

    //uint8_t *d_nonce_data;
    uint8_t* h_nonce_data = new uint8_t[NONCE_SIZE_BYTES];

    ///////////////////////////////////////////////////////////////////////////


    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, deviceId);

    std::cout << "BTCW GPU MINER RELEASE v26.5.3 - May 8 2025" << std::endl;

    std::cout << "Max threads per block: " << deviceProps.maxThreadsPerBlock << std::endl;


    // For a 1D grid:
    printf("Max grid size in X: %d\n", deviceProps.maxGridSize[0]); // x-dimension
    printf("Max grid size in Y: %d\n", deviceProps.maxGridSize[1]); // y-dimension
    printf("Max grid size in Z: %d\n", deviceProps.maxGridSize[2]); // z-dimension


    // Allocate memory on the device
    //hipMalloc(&d_gpu_num, 1);
    hipDeviceptr_t d_gpu_num;
    hipMalloc(&d_gpu_num, 1);


    // Allocate memory on the device
    //hipMalloc(&d_ctx_data, CTX_SIZE_BYTES);
    hipDeviceptr_t d_ctx_data;
    hipMalloc(&d_ctx_data, CTX_SIZE_BYTES);


    // Allocate memory on the device
    //hipMalloc(&d_key_data, KEY_SIZE_BYTES);
    hipDeviceptr_t d_key_data;
    hipMalloc(&d_key_data, KEY_SIZE_BYTES);


    // Allocate memory on the device
    //hipMalloc(&d_hash_no_sig_data, HASH_NO_SIG_SIZE_BYTES);
    hipDeviceptr_t d_hash_no_sig_data;
    hipMalloc(&d_hash_no_sig_data, HASH_NO_SIG_SIZE_BYTES);


    // Allocate memory on the device
    //hipMalloc(&d_nonce_data, NONCE_SIZE_BYTES);
    hipDeviceptr_t d_nonce_data;
    hipMalloc(&d_nonce_data, NONCE_SIZE_BYTES);



    hipStream_t stream, kernel_stream;
    hipStreamCreateWithFlags(&kernel_stream, 0);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);


    //===========================================KERNEL======================================================
    // We are starting the KERNEL with NO DATA - This is intentional, data will be given to it on the fly from the BTCW node.
    //__global__ void cuda_miner(BYTE* d_gpu_num, BYTE* key_data, BYTE* ctx_data, BYTE* hash_no_sig_in, BYTE* nonce4host )


    void* args[] = {
        &d_gpu_num,
        &d_key_data,
        &d_ctx_data,
        &d_hash_no_sig_data,
        &d_nonce_data
    };

    res = hipModuleLaunchKernel(
        cuFunction,
        128, 1, 1,     // Grid dimensions
        256, 1, 1,     // Block dimensions
        0,             // Shared memory size
        kernel_stream, // Stream
        args,          // Kernel arguments
        nullptr        // Extra (usually null)
    );

    if (res != hipSuccess) {
        const char* errStr = nullptr;
        hipDrvGetErrorString(res, &errStr);
        std::cerr << "hipModuleLaunchKernel failed: " << (errStr ? errStr : "Unknown error") << std::endl;
    }
    //=================================================================================================================


    // Map shared memory into the process's address space
    SharedData* shared_data;


#ifdef _WIN32
    // Windows code here
    // Create or open a named shared memory region
    HANDLE hMapFile = CreateFileMappingA(
        INVALID_HANDLE_VALUE,    // Use system paging file
        NULL,                    // Default security
        PAGE_READWRITE,          // Read/write access
        0,                       // Maximum object size (high-order DWORD)
        sizeof(SharedData),      // Maximum object size (low-order DWORD)
        SHM_NAME                 // Name of mapping object
    );

    if (hMapFile == NULL) {
        std::cerr << "Could not create file mapping object: " << GetLastError() << std::endl;
        return;
    }

    // Map the view of the file into the process's address space
    shared_data = (SharedData*)MapViewOfFile(
        hMapFile,            // Handle to mapping object
        FILE_MAP_ALL_ACCESS, // Read/write permission
        0,
        0,
        sizeof(SharedData)
    );

    if (shared_data == NULL) {
        std::cerr << "Could not map view of file: " << GetLastError() << std::endl;
        CloseHandle(hMapFile);
        return;
    }

#else
    // Linux/POSIX code here

    shm_unlink(SHM_NAME);

    // Open shared memory
    int shm_fd = shm_open(SHM_NAME, O_CREAT | O_RDWR, 0666);
    if (shm_fd == -1) {
        std::cerr << "Error creating shared memory" << std::endl;
        return;
    }

    // Set the size of the shared memory region
    if (ftruncate(shm_fd, sizeof(SharedData)) == -1) {
        std::cerr << "Error setting size of shared memory" << std::endl;
        return;
    }

    // Map shared memory into the process's address space
    shared_data = (SharedData*)mmap(NULL, sizeof(SharedData), PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
    if (shared_data == MAP_FAILED) {
        std::cerr << "Error mapping shared memory" << std::endl;
        return;
    }

#endif


    uint256 hash_no_sig;

    uint64_t* p_data = (uint64_t*)shared_data->data;


    // Cast to the volatile pointer to ensure we don't optimize reads/writes
    volatile SharedData* mapped_data = (volatile SharedData*)shared_data;

    // Tell the miner which GPU number it is
    //hipMemcpyAsync(d_gpu_num, h_gpu_num, 1, hipMemcpyHostToDevice, stream);
    // Async copy device -> host
    hipMemcpyHtoDAsync(d_gpu_num, h_gpu_num, 1, stream);

    uint32_t throttle = 0x0;

    while (true)
    {

        if ((throttle % 0x3) == 0)
        {

            //Host update the data, send it to the GPU
            printf("STAGE2 BLOCK DATA - CPU SIDE\n");


            // Data set from BTCW node
            memcpy(&h_key_data[0], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[0])), 32);


            memcpy(&h_ctx_data[0], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[32])), 8);
            memcpy(&h_ctx_data[8], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[40])), 8);
            memcpy(&h_ctx_data[16], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[48])), 8);
            memcpy(&h_ctx_data[24], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[56])), 8);

            memcpy(&h_ctx_data[32], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[64])), 8);
            memcpy(&h_ctx_data[40], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[72])), 8);
            memcpy(&h_ctx_data[48], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[80])), 8);
            memcpy(&h_ctx_data[56], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[88])), 8);
            memcpy(&h_ctx_data[64], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[96])), 8);
            memcpy(&h_ctx_data[72], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[104])), 8);
            memcpy(&h_ctx_data[80], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[112])), 8);
            memcpy(&h_ctx_data[88], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[120])), 8);
            memcpy(&h_ctx_data[96], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[128])), 8);
            memcpy(&h_ctx_data[104], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[136])), 8);
            memcpy(&h_ctx_data[112], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[144])), 8);
            memcpy(&h_ctx_data[120], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[152])), 8);
            memcpy(&h_ctx_data[128], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[160])), 8);
            memcpy(&h_ctx_data[136], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[168])), 8);
            memcpy(&h_ctx_data[144], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[176])), 8);

            memcpy(&h_ctx_data[152], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[184])), 4);
            memcpy(&h_ctx_data[156], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[188])), 4);


            memcpy(&h_hash_no_sig_data[0], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[192])), 8);
            memcpy(&h_hash_no_sig_data[8], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[200])), 8);
            memcpy(&h_hash_no_sig_data[16], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[208])), 8);
            memcpy(&h_hash_no_sig_data[24], const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->data[216])), 8);


            // Copy the modified data from the host back to the GPU asynchronously  
            //hipMemcpyAsync(d_ctx_data, h_ctx_data, CTX_SIZE_BYTES, hipMemcpyHostToDevice, stream);
            hipMemcpyHtoDAsync(d_ctx_data, h_ctx_data, CTX_SIZE_BYTES, stream);

            //hipMemcpyAsync(d_key_data, h_key_data, KEY_SIZE_BYTES, hipMemcpyHostToDevice, stream);
            hipMemcpyHtoDAsync(d_key_data, h_key_data, KEY_SIZE_BYTES, stream);

            //hipMemcpyAsync(d_hash_no_sig_data, h_hash_no_sig_data, HASH_NO_SIG_SIZE_BYTES, hipMemcpyHostToDevice, stream);   
            hipMemcpyHtoDAsync(d_hash_no_sig_data, h_hash_no_sig_data, HASH_NO_SIG_SIZE_BYTES, stream);

        }

        throttle++;

        //hipMemcpyAsync(const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->nonce)), d_nonce_data, NONCE_SIZE_BYTES, hipMemcpyDeviceToHost, stream); 
        hipMemcpyDtoHAsync(const_cast<void*>(reinterpret_cast<const volatile void*>(&shared_data->nonce)), d_nonce_data, NONCE_SIZE_BYTES, stream);

        Sleep(500);

    }



    // Cleanup
    hipFree(d_gpu_num);
    delete[] h_gpu_num;

    hipFree(d_ctx_data);
    delete[] h_ctx_data;

    hipFree(d_key_data);
    delete[] h_key_data;

    hipFree(d_hash_no_sig_data);
    delete[] h_hash_no_sig_data;

    hipFree(d_nonce_data);
    delete[] h_nonce_data;


    hipStreamDestroy(stream);
    hipStreamDestroy(kernel_stream);

    return 0;
}

